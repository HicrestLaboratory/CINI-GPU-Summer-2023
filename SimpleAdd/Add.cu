
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
// Kernel function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{
  for (int i = 0; i < n; i++)
    y[i] = x[i] + y[i];
}

int main(void)
{
  int N = 1<<20;
  float *x, *y;

  // Create the event for getting the time
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // Allocate Unified Memory accessible from CPU or GPU
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // Record the event
  hipEventRecord(start);
  // Run kernel on 1M elements on the GPU
  add<<<1, 1>>>(N, x, y);
  // Wait for GPU to finish before accessing on host
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  // Altarntivelly you can you cudaDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;

  // Measure the time recorded
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  std::cout << "Kernel Time (ms): " << milliseconds << std::endl;

  hipEventDestroy(start);
  hipEventDestroy(stop);
  // Free memory
  hipFree(x);
  hipFree(y);
  
  return 0;
}
